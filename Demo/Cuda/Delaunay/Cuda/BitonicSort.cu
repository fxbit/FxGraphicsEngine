#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

#include "includes.h"

extern "C"  {	
	// Device code
	__global__ void Triangulation(ThreadInfo* threadInfoArray, const RegionInfo *regionInfoArray, const ThreadParam param, const int RegionsNum)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i < RegionsNum){
            
		}
	}
}
