#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>


#define uint unsigned int
#define DATA_TYPE float2

#include "includes.h"
#include "IOBuffer.h"
#include "HalfEdgeUtils.h"
#include "FaceUtils.h"
#include "TriangleUtils.h"
#include "BoundaryUtils.h"

__device__
void Init(const DATA_TYPE   *VertexList, 
          HalfEdge          *HEList,
          BoundaryNode      *BoundaryList,
          Face              *FaceList,
          ThreadInfo        *threadInfo)
{
    uint bn1_ID, bn2_ID, bn3_ID;
    BoundaryNode bn1,bn2,bn3;
    HalfEdge he1,he2,he3;
    uint3 heID;
    uint offset =  threadInfo->offsetVertexList;
    bool ccw;
    
    // get the 3 first points and create a triangle
    // create the first triangle
    CreateTriangle<true>(VertexList, HEList, threadInfo, 
                         offset, offset+1 , offset+2 ,// the 3 first vertex
                         &he1, &he2, &he3,
                         &heID,
                         &ccw);
    
    // create a new face that start from HalfEdge1ID
    CreateFace(HEList,
               FaceList,
               heID.x,
               threadInfo);
    
    // create and link the boundary list   
    {
        
        // read again the face that created from the 0,1,2 vertex ... CCW problems
        bn1_ID = InitNewBoundaryNode(VertexList, HEList, threadInfo,
                                     heID.x, &bn1);			            //  init the root bn
        bn2_ID = InitNewBoundaryNode(VertexList, HEList, threadInfo,
                                     he1.nextEdgeID, &bn2);	            //  init the second bn
        he2    = HEList[he1.nextEdgeID];                 				//  move to the next he
        bn3_ID = InitNewBoundaryNode(VertexList, HEList, threadInfo,
                                     he2.nextEdgeID, &bn3);	            //  init the third bn
        he3    = HEList[he2.nextEdgeID];					            //  move to the next he

        // set the root bn to 1
        threadInfo->boundaryNodeRootID = bn1_ID;

        // link the root with the next node
        bn1.NextNodeID = bn2_ID;
        bn2.PrevNodeID = bn1_ID;

        // link the root with the next node
        bn2.NextNodeID = bn3_ID;
        bn3.PrevNodeID = bn2_ID;

        // link the first with the last one
        bn3.NextNodeID = bn1_ID;
        bn1.PrevNodeID = bn3_ID;


        // store the new bn
        SetBoundaryNode(BoundaryList, &bn1, bn1_ID,  threadInfo);
        SetBoundaryNode(BoundaryList, &bn2, bn2_ID,  threadInfo);
        SetBoundaryNode(BoundaryList, &bn3, bn3_ID,  threadInfo);

    }

}


// ---------------------------------------------------------------------------------

extern "C"
__global__ void Triangulation(const DATA_TYPE   *VertexList,
                              HalfEdge          *HEList,
                              BoundaryNode      *BoundaryList,
                              Face              *FaceList,
                              ThreadInfo        *threadInfoArray,
                              const RegionInfo  *regionInfoArray,
                              const ThreadParam  param,
                              const int          RegionsNum)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= RegionsNum)
        return;
    
    ThreadInfo threadInfo;
    RegionInfo regionInfo           = regionInfoArray[i];
    
    threadInfo.threadID             =  i;
    threadInfo.offsetFaceList		=  i * param.maxFacesPerThread;
    threadInfo.offsetHalfEdgeList	=  i * param.maxHalfEdgePerThread;
    threadInfo.offsetVertexList		=  regionInfo.VertexOffset;
    threadInfo.offsetBoundaryList	=  i * param.maxBoundaryNodesPerThread;
    threadInfo.offsetDNStack		=  i * MAX_FACE_CORRECTIONS;

    threadInfo.lastFaceID			=  make_uint2(0,threadInfo.offsetFaceList); 	// no face yet
    threadInfo.lastHalfEdgeID		=  threadInfo.offsetHalfEdgeList; 	// no he yet
    threadInfo.lastBoundaryNodeID	=  0; 	// no bn yet
    threadInfo.boundaryNodeRootID	=  0; 	// set the node root to unset;
    threadInfo.endDNOfStack 		=  0; 	// no DN in stack yet
    threadInfo.startDNOfStack 		=  0; 	// no DN in stack yet
    threadInfo.numDNinStack			=  0; 	// no DN in stack yet

    // init the triangulation by create the first triangle
    Init(VertexList, 
         HEList,
         BoundaryList,
         FaceList,
         &threadInfo);
    
    // save the results back to array
    threadInfoArray[i] = threadInfo;

}
